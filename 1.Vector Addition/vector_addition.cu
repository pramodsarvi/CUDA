#include<hip/hip_runtime.h>

#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<assert.h>
#include<iostream>
using namespace std;

__global__ void vectorAdd(int* a,int* b,int* c, int n)
{
    int thread_id = (blockIdx.x*blockDim.x) + threadIdx.x;
    if(thread_id<n)
    {
        c[thread_id] = a[thread_id] + b[thread_id];
    }
}

void verify_result(int *a, int *b, int *c, int N) {
  for (int i = 0; i < N; i++) {
    assert(c[i] == a[i] + b[i]);
  }
}



int main() {
  // Array size of 2^16 (65536 elements)
  constexpr int N = 1 << 26;
  size_t bytes = sizeof(int) * N;

  // Vectors for holding the host-side (CPU-side) data
  int *h_a, *h_b, *h_c;

  // Allocate pinned memory
  hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
  hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
  hipHostMalloc(&h_c, bytes, hipHostMallocDefault);

  // Initialize random numbers in each array
  for(int i = 0; i < N; i++){
    h_a[i] = rand() % 100;
    h_b[i] = rand() % 100;
  }
  
  // Allocate memory on the device
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Copy data from the host to the device (CPU -> GPU)
  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  // Threads per CTA (1024 threads per CTA)
  int NUM_THREADS = 1 << 10;

  // CTAs per Grid
  // We need to launch at LEAST as many threads as we have elements
  // This equation pads an extra CTA to the grid if N cannot evenly be divided
  // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

  // Launch the kernel on the GPU
  // Kernel calls are asynchronous (the CPU program continues execution after
  // call, but no necessarily before the kernel finishes)
  vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

  // Copy sum vector from device to host
  // cudaMemcpy is a synchronous operation, and waits for the prior kernel
  // launch to complete (both go to the default stream in this case).
  // Therefore, this cudaMemcpy acts as both a memcpy and synchronization
  // barrier.
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  // Check result for errors
  verify_result(h_a, h_b, h_c, N);

  // Free pinned memory
  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_c);

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  std::cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}